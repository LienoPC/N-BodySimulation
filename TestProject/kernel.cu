#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <math.h>
#include <stdexcept>
#include "constants.h"


__device__ float4 bodyInteractions(float4 bi, float4 bj, float4 ai) {
	// Compute r_ij position vector of i from j
	float3 r_ij;
	r_ij.x = bj.x - bi.x;
	r_ij.y = bj.y - bi.y;
	r_ij.z = bj.z - bi.z;

	// Sqrt of ||r_ij||^2 + EPS^2
	float distSqrt = r_ij.x * r_ij.x + r_ij.y * r_ij.y + r_ij.z * r_ij.z + EPS2;

	// distSqrt^3
	float invDenom = 1.0f / sqrtf(distSqrt * distSqrt * distSqrt);

	// factor = m_j * invDenom
	float factor = bj.w * invDenom;
	ai.x += r_ij.x * factor;
	ai.y += r_ij.y * factor;
	ai.z += r_ij.z * factor;

	return ai;	
}


__device__ float3 bodyInteractions_float3(float4 bi, float4 bj, float3 ai) {
	// Compute r_ij position vector of i from j
	float3 r_ij;
	r_ij.x = bj.x - bi.x;
	r_ij.y = bj.y - bi.y;
	r_ij.z = bj.z - bi.z;

	// Sqrt of ||r_ij||^2 + EPS^2
	float distSqrt = r_ij.x * r_ij.x + r_ij.y * r_ij.y + r_ij.z * r_ij.z + EPS2;

	// distSqrt^3
	float invDenom = 1.0f / sqrtf(distSqrt * distSqrt * distSqrt);

	// factor = m_j * invDenom
	float factor = bj.w * invDenom;
	ai.x += r_ij.x * factor;
	ai.y += r_ij.y * factor;
	ai.z += r_ij.z * factor;

	return ai;
}


__device__ float4 tile_interaction(float4 myBody, float4 accel, int tileFirstIdx, int tileWidthFactor, int N) {
	int i;
	extern __shared__  __align__(16) float4 shBodies[];
	
	// Unrolling loop to increase ILP
	#pragma unroll
	for (i = 0; i < blockDim.x * tileWidthFactor && i < N - tileFirstIdx; i++) {
		accel = bodyInteractions(myBody, shBodies[i], accel);
	}
	return accel;
}

__device__ float3 tile_interaction_float3(float4 myBody, float3 accel) {
	int i;
	extern __shared__  __align__(16) float4 shBodies[];

	// Unrolling loop to increase ILP
#pragma unroll
	for (i = 0; i < blockDim.x; i++) {
		accel = bodyInteractions_float3(myBody, shBodies[i], accel);
	}
	return accel;
}


__global__ void kernel(float4* globalX, float4* globalA, float4* globalV, int N, int tileWidthFactor) {
	extern __shared__  __align__(16) float4 shBodies[];
	float4 myBody, myNewBody; // Position (x, y, z) and weight (w)
	float4 myNewVel;
	float4 myNewAccel = { 0.0f, 0.0f, 0.0f, 0.0f };
	int tileWidth, globalIdx, sharedIdx;
	
	// We use 1D blocks because each thread in a block computes the interactions between its body and each other body serially,
	// fetching the descriptions of the other bodies from shared memory after they've been loaded.
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= N) {
		return;
	}

	myNewBody = globalX[tid];
	myNewVel = globalV[tid];
	tileWidth = blockDim.x * tileWidthFactor;

	// Each tile is of size (blockDim.x, blockDim.x * tileWidthFactor)
	for (int i = 0, int tile = 0; i < N; i += tileWidth, tile++) {
		// Load data into shared memory if within bounds
		for (int j = 0; j < tileWidthFactor; j++) {
			sharedIdx = blockDim.x * j + threadIdx.x;
			globalIdx = tile * tileWidth + blockDim.x * j + threadIdx.x;

			if (globalIdx < N) { // Make sure we avoid out of bounds accesses if the last tile is smaller
				shBodies[sharedIdx] = globalX[globalIdx];
			}
		}
		__syncthreads();

		// Compute interactions
		myNewAccel = tile_interaction(myNewBody, myNewAccel, tile * tileWidth, tileWidthFactor, N);
		__syncthreads();
	}

	// Update the body's velocity by half a step
	myNewVel.x += 0.5f * DT * myNewAccel.x; 
	myNewVel.y += 0.5f * DT * myNewAccel.y;
	myNewVel.z += 0.5f * DT * myNewAccel.z;

	// Update the body's position by a full-step
	myNewBody.x += DT * myNewVel.x;
	myNewBody.y += DT * myNewVel.y;
	myNewBody.z += DT * myNewVel.z;

	// Store the integration result in global memory
	globalX[tid] = myNewBody;
	globalV[tid] = myNewVel;   
	globalA[tid] = myNewAccel;  
}


__device__ void warpReduce(volatile float4* shMem, float4 accel, int sid) {

	shMem[sid].x += shMem[sid + 16].x;
	shMem[sid].y += shMem[sid + 16].y;
	shMem[sid].z += shMem[sid + 16].z;


	shMem[sid].x += shMem[sid + 8].x;
	shMem[sid].y += shMem[sid + 8].y;
	shMem[sid].z += shMem[sid + 8].z;


	shMem[sid].x += shMem[sid + 4].x;
	shMem[sid].y += shMem[sid + 4].y;
	shMem[sid].z += shMem[sid + 4].z;


	shMem[sid].x += shMem[sid + 2].x;
	shMem[sid].y += shMem[sid + 2].y;
	shMem[sid].z += shMem[sid + 2].z;

	shMem[sid].x += shMem[sid + 1].x;
	shMem[sid].y += shMem[sid + 1].y;
	shMem[sid].z += shMem[sid + 1].z;
	//shMem[sid] = accel;

}


// Embarassingly parallel kernel version
__global__ void kernel_reduction(float4* globalX, float4* reduceMatrix, int N) {
	extern __shared__  __align__(16) float4 shMem[];
	float4 baseBody; // Position (x, y, z) and weight (w)
	float4 otherBody;
	float4 myNewAccel = { 0.0f, 0.0f, 0.0f, 0.0f };


	//float4 reduceBody;
	// We are working in this algorithm in an NxN grid of threads, in which every thread computes one single interaction
	// that then will be reduced and combined row-wise with all other acceleration over the same body
	int tidX = blockIdx.x * blockDim.x + threadIdx.x; // Defines the body with which the computation should be
	int tidY = blockIdx.y * blockDim.y + threadIdx.y; // Defines the effective body for which we are computing the interacion and over which will be executed the reduction
	// Shared memory id
	int sid = threadIdx.y * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	if (tidX >= N || tidY >= N) {

		return;
	}
	baseBody = globalX[tidY];
	otherBody = globalX[tidX];
	
	// Compute the single interaction
	myNewAccel = bodyInteractions(baseBody, otherBody, myNewAccel);

	// Load into shared memory along X dimension
	shMem[sid] = myNewAccel;

	// Wait for all threads completion
	__syncthreads();

	// Sequential Addressing Reduction
	// 4-way bank conflict
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			shMem[sid].x += shMem[sid + s].x;
			shMem[sid].y += shMem[sid + s].y;
			shMem[sid].z += shMem[sid + s].z;
		}
		__syncthreads();
	}

	// At this point, we have the sum of all blocks X-wise computed interactions
	// We need now to sum all the blocks over the X-axis for each body
	if (threadIdx.x == 0) {
		reduceMatrix[tidY * gridDim.x + blockIdx.x] = shMem[sid];
	}
}


// Embarassingly parallel kernel version
__global__ void kernel_reduction_float3(float4* globalX, float3* reduceMatrix, int N) {
	extern __shared__  float3 shMem3[];
	float4 baseBody; // Position (x, y, z) and weight (w)
	float4 otherBody;
	float3 myNewAccel = { 0.0f, 0.0f, 0.0f};

	// We are working in this algorithm in an NxN grid of threads, in which every thread computes one single interaction
	// that then will be reduced and combined row-wise with all other acceleration over the same body
	int tidX = blockIdx.x * blockDim.x + threadIdx.x; // Defines the body with which the computation should be
	int tidY = blockIdx.y * blockDim.y + threadIdx.y; // Defines the effective body for which we are computing the interacion and over which will be executed the reduction
	// Shared memory id
	int sid = threadIdx.y * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	if (tidX >= N || tidY >= N) {

		return;
	}
	baseBody = globalX[tidY];
	otherBody = globalX[tidX];

	// Compute the single interaction
	myNewAccel = bodyInteractions_float3(baseBody, otherBody, myNewAccel);

	// Load into shared memory along X dimension
	shMem3[sid] = myNewAccel;

	// Wait for all threads completion
	__syncthreads();

	// Loop unrolling, knowing that we are using block of 32 threads
	if (tid < 16) {
		shMem3[sid].x += shMem3[sid + 16].x;
		shMem3[sid].y += shMem3[sid + 16].y;
		shMem3[sid].z += shMem3[sid + 16].z;
	}
	__syncthreads();
	if (tid < 8) {
		shMem3[sid].x += shMem3[sid + 8].x;
		shMem3[sid].y += shMem3[sid + 8].y;
		shMem3[sid].z += shMem3[sid + 8].z;
	}
	__syncthreads();
	if (tid < 4) {
		shMem3[sid].x += shMem3[sid + 4].x;
		shMem3[sid].y += shMem3[sid + 4].y;
		shMem3[sid].z += shMem3[sid + 4].z;
	}
	__syncthreads();
	if (tid < 2) {
		shMem3[sid].x += shMem3[sid + 2].x;
		shMem3[sid].y += shMem3[sid + 2].y;
		shMem3[sid].z += shMem3[sid + 2].z;
	}
	__syncthreads();
	if (tid < 1) {
		shMem3[sid].x += shMem3[sid + 1].x;
		shMem3[sid].y += shMem3[sid + 1].y;
		shMem3[sid].z += shMem3[sid + 1].z;
	}

	__syncthreads();

	// At this point, we have the sum of all blocks X-wise computed interactions
	// We need now to sum all the blocks over the X-axis for each body
	if (threadIdx.x == 0) {
		reduceMatrix[tidY * gridDim.x + blockIdx.x] = shMem3[sid];
	}
}


// Embarassingly parallel kernel version with first add during load for reduction
__global__ void kernel_reduction_fadl(float4* globalX, float4* reduceMatrix, int N) {
	extern __shared__  __align__(16) float4 shMem[];
	//float4 baseBody; // Position (x, y, z) and weight (w)
	//float4 otherBody;
	float myNewAccel1x = 0.0f;
	float myNewAccel1y = 0.0f;
	float myNewAccel1z = 0.0f;
	float4 myNewAccel1 = { 0.0f, 0.0f, 0.0f, 0.0f };
	float4 myNewAccel2 = { 0.0f, 0.0f, 0.0f, 0.0f };

	// This version of the algorithm uses half of the threads
	int tidX = blockIdx.x * (blockDim.x*2) + threadIdx.x; // Defines the body with which the computation should be
	int tidY = blockIdx.y * blockDim.y + threadIdx.y; // Defines the effective body for which we are computing the interacion and over which will be executed the reduction
	// Shared memory id
	int sid = threadIdx.y * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	if (tidX >= N || tidY >= N) {

		return;
	}
	myNewAccel1 = bodyInteractions(globalX[tidY], globalX[tidX], myNewAccel1);
	myNewAccel2 = bodyInteractions(globalX[tidY], globalX[tidX + blockDim.x], myNewAccel2);


	myNewAccel1x = myNewAccel1.x;
	myNewAccel1y = myNewAccel1.y;
	myNewAccel1z = myNewAccel1.z;

	// Load into shared memory along X dimension
	myNewAccel1x += myNewAccel2.x;
	myNewAccel1y += myNewAccel2.y;
	myNewAccel1z += myNewAccel2.z;
	shMem[sid] = myNewAccel1;
	// Wait for all threads completion
	__syncthreads();

	// Loop unrolling, knowing that we are using block of 32 threads (in this case, halved)
	// Each reduction step produces a 4-way bank conflict
	if (tid < 8) {
		myNewAccel1x += shMem[sid + 8].x;
		myNewAccel1y += shMem[sid + 8].y;
		myNewAccel1z += shMem[sid + 8].z;
		shMem[sid].x = myNewAccel1x;
		shMem[sid].y = myNewAccel1y;
		shMem[sid].z = myNewAccel1z;

	}
	__syncthreads();
	if (tid < 4) {
		myNewAccel1x += shMem[sid + 4].x;
		myNewAccel1y += shMem[sid + 4].y;
		myNewAccel1z += shMem[sid + 4].z;
		shMem[sid].x = myNewAccel1x;
		shMem[sid].y = myNewAccel1y;
		shMem[sid].z = myNewAccel1z;
	}
	__syncthreads();
	if (tid < 2) {
		myNewAccel1x += shMem[sid + 2].x;
		myNewAccel1y += shMem[sid + 2].y;
		myNewAccel1z += shMem[sid + 2].z;
		shMem[sid].x = myNewAccel1x;
		shMem[sid].y = myNewAccel1y;
		shMem[sid].z = myNewAccel1z;
	}
	__syncthreads();
	if (tid < 1) {
		myNewAccel1x += shMem[sid + 1].x;
		myNewAccel1y += shMem[sid + 1].y;
		myNewAccel1z += shMem[sid + 1].z;
		shMem[sid].x = myNewAccel1x;
		shMem[sid].y = myNewAccel1y;
		shMem[sid].z = myNewAccel1z;
	}
	
	__syncthreads();

	// At this point, we have the sum of all blocks X-wise computed interactions
	// We need now to sum all the blocks over the X-axis for each body
	if (threadIdx.x == 0) {
		reduceMatrix[tidY * gridDim.x + blockIdx.x] = shMem[sid];
	}
}


// Embarassingly parallel kernel version with first add during load for reduction
__global__ void kernel_reduction_fadl4(float4* globalX, float4* reduceMatrix, int N) {
	extern __shared__  __align__(16) float4 shMem[];

	float4 myNewAccel1 = { 0.0f, 0.0f, 0.0f, 0.0f };
	float4 myNewAccel2 = { 0.0f, 0.0f, 0.0f, 0.0f };
	float4 myNewAccel3 = { 0.0f, 0.0f, 0.0f, 0.0f };
	float4 myNewAccel4 = { 0.0f, 0.0f, 0.0f, 0.0f };

	// This version of the algorithm uses half of the threads
	int tidX = blockIdx.x * (blockDim.x * 4) + threadIdx.x; // Defines the body with which the computation should be done
	int tidY = blockIdx.y * blockDim.y + threadIdx.y; // Defines the effective body for which we are computing the interacion and over which will be executed the reduction
	// Shared memory id
	int sid = threadIdx.y * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	if (tidX >= N || tidY >= N) {

		return;
	}
	myNewAccel1 = bodyInteractions(globalX[tidY], globalX[tidX], myNewAccel1);
	myNewAccel2 = bodyInteractions(globalX[tidY], globalX[tidX + blockDim.x], myNewAccel2);
	myNewAccel3 = bodyInteractions(globalX[tidY], globalX[tidX + blockDim.x * 2], myNewAccel3);
	myNewAccel4 = bodyInteractions(globalX[tidY], globalX[tidX + blockDim.x * 3], myNewAccel4);

	// Compute the add operation
	myNewAccel1.x += myNewAccel2.x + myNewAccel3.x + myNewAccel4.x;
	myNewAccel1.y += myNewAccel2.y + myNewAccel3.y + myNewAccel4.y;
	myNewAccel1.z += myNewAccel2.z + myNewAccel3.z + myNewAccel4.z;
	shMem[sid] = myNewAccel1;
	// Wait for all threads completion
	__syncthreads();

	// Loop unrolling, knowing that we are using block of 32 threads (in this case, halved)
	// Each reduction step produces a 4-way bank conflict
	if (tid < 8) {
		myNewAccel1.x += shMem[sid + 8].x;
		myNewAccel1.y += shMem[sid + 8].y;
		myNewAccel1.z += shMem[sid + 8].z;
		shMem[sid] = myNewAccel1;
	}

	__syncthreads();
	if (tid < 4) {
		myNewAccel1.x += shMem[sid + 4].x;
		myNewAccel1.y += shMem[sid + 4].y;
		myNewAccel1.z += shMem[sid + 4].z;
		shMem[sid] = myNewAccel1;
	}

	__syncthreads();
	if (tid < 2) {
		myNewAccel1.x += shMem[sid + 2].x;
		myNewAccel1.y += shMem[sid + 2].y;
		myNewAccel1.z += shMem[sid + 2].z;
		shMem[sid] = myNewAccel1;
	}

	__syncthreads();
	if (tid < 1) {
		myNewAccel1.x += shMem[sid + 1].x;
		myNewAccel1.y += shMem[sid + 1].y;
		myNewAccel1.z += shMem[sid + 1].z;
		shMem[sid] = myNewAccel1;
	}

	__syncthreads();

	// At this point, we have the sum of all blocks X-wise computed interactions
	// We need now to sum all the blocks over the X-axis for each body
	if (threadIdx.x == 0) {
		reduceMatrix[tidY * gridDim.x + blockIdx.x] = shMem[sid];
	}
}


__global__ void inter_block_reduction(float4* globalX, float4* globalA, float4* globalV, float4* reduceMatrix, int N, int numBlocks) {
	int tidY = blockIdx.x * blockDim.x + threadIdx.x; // Each thread handles one body
	if (tidY >= N) return;

	float4 myNewAccel = { 0.0f, 0.0f, 0.0f, 0.0f };
	float4 myNewVel = globalV[tidY];
	float4 baseBody = globalX[tidY];
	// Reduce over all block contributions for this body
	for (int block = 0; block < numBlocks; block++) {
		int idx = tidY * numBlocks + block; // Flattened index in reduceMatrix
		//printf("InterBlockReduction; reduceMatrix[%d][%d] = %f\n", tidY, block, reduceMatrix[idx]);
		myNewAccel.x += reduceMatrix[idx].x;
		myNewAccel.y += reduceMatrix[idx].y;
		myNewAccel.z += reduceMatrix[idx].z;
	}

	// Update the body's velocity by half a step
	myNewVel.x += 0.5 * DT * myNewAccel.x;
	myNewVel.y += 0.5 * DT * myNewAccel.y;
	myNewVel.z += 0.5 * DT * myNewAccel.z;

	// Update the body's position by a full-step
	baseBody.x += DT * myNewVel.x;
	baseBody.y += DT * myNewVel.y;
	baseBody.z += DT * myNewVel.z;

	// Store the integration result in global memory
	globalX[tidY] = baseBody;
	globalV[tidY] = myNewVel;
	globalA[tidY] = myNewAccel;
}


__global__ void inter_block_reduction_float3(float4* globalX, float3* globalA, float3* globalV, float3* reduceMatrix, int N, int numBlocks) {
	int tidY = blockIdx.x * blockDim.x + threadIdx.x; // Each thread handles one body
	if (tidY >= N) return;

	float3 myNewAccel = { 0.0f, 0.0f, 0.0f};
	float3 myNewVel = globalV[tidY];
	float4 baseBody = globalX[tidY];
	// Reduce over all block contributions for this body
	for (int block = 0; block < numBlocks; block++) {
		int idx = tidY * numBlocks + block; // Flattened index in reduceMatrix
		//printf("InterBlockReduction; reduceMatrix[%d][%d] = %f\n", tidY, block, reduceMatrix[idx]);
		myNewAccel.x += reduceMatrix[idx].x;
		myNewAccel.y += reduceMatrix[idx].y;
		myNewAccel.z += reduceMatrix[idx].z;
	}

	// Update the body's velocity by half a step
	myNewVel.x += 0.5 * DT * myNewAccel.x;
	myNewVel.y += 0.5 * DT * myNewAccel.y;
	myNewVel.z += 0.5 * DT * myNewAccel.z;

	// Update the body's position by a full-step
	baseBody.x += DT * myNewVel.x;
	baseBody.y += DT * myNewVel.y;
	baseBody.z += DT * myNewVel.z;

	// Store the integration result in global memory
	globalX[tidY] = baseBody;
	globalV[tidY] = myNewVel;
	globalA[tidY] = myNewAccel;
}


void simulateVisual_embParallel_fadl(hipGraphicsResource* graphic_res, float4* bodies, float4* d_accelerations, float4* d_velocity, float4* d_reduceMatrix, int N) {
	size_t size4 = sizeof(float4) * N;
	float4* d_bodies;


	// Map openGL buffer to cuda pointer
	hipGraphicsMapResources(1, &graphic_res, 0);

	// Get pointer to bodies
	hipGraphicsResourceGetMappedPointer((void**)&d_bodies, &size4, graphic_res);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	int threadsPerBlock = THREADS_PER_BLOCK;
	if (threadsPerBlock > deviceProp.maxThreadsPerBlock)
		throw std::runtime_error("threadsPerBlock is greater than the device maximum threads per block");

	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	dim3 blockDim(threadsPerBlock/2, threadsPerBlock);
	dim3 gridDim(blocksPerGrid/2, blocksPerGrid);
	size_t sharedMemSize = sizeof(float4) * threadsPerBlock/2 * threadsPerBlock;

	// Launch thread computation
	kernel_reduction_fadl << <gridDim, blockDim, sharedMemSize >> > (d_bodies, d_reduceMatrix, N);
	hipDeviceSynchronize();
	inter_block_reduction << < blocksPerGrid, threadsPerBlock >> > (d_bodies, d_accelerations, d_velocity, d_reduceMatrix, N, blocksPerGrid/2);

	hipGraphicsUnmapResources(1, &graphic_res, 0);
}


void simulateVisual_embParallel_fadl4(hipGraphicsResource* graphic_res, float4* bodies, float4* d_accelerations, float4* d_velocity, float4* d_reduceMatrix, int N) {
	size_t size4 = sizeof(float4) * N;
	float4* d_bodies;

	// Map openGL buffer to cuda pointer
	hipGraphicsMapResources(1, &graphic_res, 0);

	// Get pointer to bodies
	hipGraphicsResourceGetMappedPointer((void**)&d_bodies, &size4, graphic_res);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	int threadsPerBlock = THREADS_PER_BLOCK;
	if (threadsPerBlock > deviceProp.maxThreadsPerBlock)
		throw std::runtime_error("threadsPerBlock is greater than the device maximum threads per block");

	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	dim3 blockDim(threadsPerBlock / 4, threadsPerBlock);
	dim3 gridDim(blocksPerGrid / 4, blocksPerGrid);
	size_t sharedMemSize = sizeof(float4) * threadsPerBlock / 4 * threadsPerBlock;

	// Launch thread computation
	kernel_reduction_fadl << <gridDim, blockDim, sharedMemSize >> > (d_bodies, d_reduceMatrix, N);
	hipDeviceSynchronize();
	inter_block_reduction << < blocksPerGrid, threadsPerBlock >> > (d_bodies, d_accelerations, d_velocity, d_reduceMatrix, N, blocksPerGrid / 4);

	hipGraphicsUnmapResources(1, &graphic_res, 0);
}

void simulateVisual_embParallel(hipGraphicsResource* graphic_res, float4* bodies, float4* d_accelerations, float4* d_velocity, float4* d_reduceMatrix, int N) {
	size_t size4 = sizeof(float4) * N;
	float4* d_bodies;

	// Map openGL buffer to cuda pointer
	hipGraphicsMapResources(1, &graphic_res, 0);

	// Get pointer to bodies
	hipGraphicsResourceGetMappedPointer((void**)&d_bodies, &size4, graphic_res);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	int threadsPerBlock = THREADS_PER_BLOCK;
	if (threadsPerBlock > deviceProp.maxThreadsPerBlock)
		throw std::runtime_error("threadsPerBlock is greater than the device maximum threads per block");

	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	dim3 blockDim(threadsPerBlock, threadsPerBlock);
	dim3 gridDim(blocksPerGrid, blocksPerGrid);
	size_t sharedMemSize = sizeof(float4) * threadsPerBlock * threadsPerBlock;

	// Launch thread computation
	kernel_reduction << <gridDim, blockDim, sharedMemSize >> > (d_bodies, d_reduceMatrix, N);
	hipDeviceSynchronize();
	inter_block_reduction << < blocksPerGrid, threadsPerBlock >> > (d_bodies, d_accelerations, d_velocity, d_reduceMatrix, N, blocksPerGrid);

	hipGraphicsUnmapResources(1, &graphic_res, 0);
}

void simulateVisual_embParallel_float3(hipGraphicsResource* graphic_res, float4* bodies, float3* d_accelerations, float3* d_velocity, float3* d_reduceMatrix, int N) {
	size_t size4 = sizeof(float4) * N;
	float4* d_bodies;

	// Map openGL buffer to cuda pointer
	hipGraphicsMapResources(1, &graphic_res, 0);

	// Get pointer to bodies
	hipGraphicsResourceGetMappedPointer((void**)&d_bodies, &size4, graphic_res);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	int threadsPerBlock = THREADS_PER_BLOCK;
	if (threadsPerBlock > deviceProp.maxThreadsPerBlock)
		throw std::runtime_error("threadsPerBlock is greater than the device maximum threads per block");

	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	dim3 blockDim(threadsPerBlock, threadsPerBlock);
	dim3 gridDim(blocksPerGrid, blocksPerGrid);
	size_t sharedMemSize = sizeof(float4) * threadsPerBlock * threadsPerBlock;

	// Launch thread computation
	kernel_reduction_float3 << <gridDim, blockDim, sharedMemSize >> > (d_bodies, d_reduceMatrix, N);
	hipDeviceSynchronize();
	inter_block_reduction_float3 << < blocksPerGrid, threadsPerBlock >> > (d_bodies, d_accelerations, d_velocity, d_reduceMatrix, N, blocksPerGrid);

	hipGraphicsUnmapResources(1, &graphic_res, 0);
}


void simulateVisual(hipGraphicsResource* graphic_res, float4* bodies, float4* d_accelerations, float4* d_velocity, int N) {
	size_t size4 = sizeof(float4) * N;
	float4* d_bodies;
	// Map openGL buffer to cuda pointer
	hipGraphicsMapResources(1, &graphic_res, 0);

	// Get pointer to bodies
	hipGraphicsResourceGetMappedPointer((void**)&d_bodies, &size4, graphic_res);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	int threadsPerBlock = THREADS_PER_BLOCK;
	if (threadsPerBlock > deviceProp.maxThreadsPerBlock)
		throw std::runtime_error("threadsPerBlock is greater than the device maximum threads per block.");

	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	size_t sharedMemSize = sizeof(float4) * threadsPerBlock * TILE_WIDTH_FACTOR;

	if (sharedMemSize > deviceProp.sharedMemPerBlock || sharedMemSize * blocksPerGrid > deviceProp.sharedMemPerMultiprocessor * deviceProp.multiProcessorCount) {
		throw std::runtime_error("Shared memory request too large.");
	}

	kernel << <blocksPerGrid, threadsPerBlock, sharedMemSize >> > (d_bodies, d_accelerations, d_velocity, N, TILE_WIDTH_FACTOR);
	hipDeviceSynchronize();

	hipGraphicsUnmapResources(1, &graphic_res, 0);
}

void simulate(float4* d_bodies, float4* d_accelerations, float4* d_velocity, int N) {
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	int threadsPerBlock = THREADS_PER_BLOCK;
	if (threadsPerBlock > deviceProp.maxThreadsPerBlock)
		throw std::runtime_error("threadsPerBlock is greater than the device maximum threads per block.");

	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	size_t sharedMemSize = sizeof(float4) * threadsPerBlock * TILE_WIDTH_FACTOR;

	if (sharedMemSize > deviceProp.sharedMemPerBlock || sharedMemSize * blocksPerGrid > deviceProp.sharedMemPerMultiprocessor * deviceProp.multiProcessorCount) {
		throw std::runtime_error("Shared memory request too large.");
	}

	kernel <<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_bodies, d_accelerations, d_velocity, N, TILE_WIDTH_FACTOR);
	hipDeviceSynchronize();
}
